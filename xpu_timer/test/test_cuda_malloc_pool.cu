// Copyright 2024 The DLRover Authors. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>

void cudaMallocWrapper(void** devPtr, size_t size) {
  hipError_t err = hipMalloc(devPtr, size);

  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  printf("Success");
}

int main() {
  int device = 0;
  hipSetDevice(device);

  float* devPtr1;
  cudaMallocWrapper((void**)&devPtr1, sizeof(float) * 100);

  hipMemPoolProps poolProps = {};
  poolProps.allocType = hipMemAllocationTypePinned;
  poolProps.location.id = device;
  poolProps.location.type = hipMemLocationTypeDevice;

  hipMemPool_t pool;
  hipError_t err = hipMemPoolCreate(&pool, &poolProps);

  if (err != hipSuccess) {
    std::cerr << "Failed to create CUDA memory pool: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }

  void* devPtr = NULL;
  size_t size = 1024 * sizeof(int);
  hipError_t cudaStatus = hipMallocFromPoolAsync(&devPtr, size, pool, 0);
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipMallocFromPoolAsync failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    return 1;
  }

  int* hostPtr = new int[1024];
  hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);

  hipFree(devPtr);
  hipMemPoolDestroy(pool);
  delete[] hostPtr;

  return 1;
}
