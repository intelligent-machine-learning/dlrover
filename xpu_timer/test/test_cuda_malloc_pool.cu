#include <hip/hip_runtime.h>
#include <stdio.h>

#include <iostream>

void cudaMallocWrapper(void **devPtr, size_t size) {
  hipError_t err = hipMalloc(devPtr, size);

  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  printf("Success");
}

int main() {
  int device = 0;
  hipSetDevice(device);

  float *devPtr1;
  cudaMallocWrapper((void **)&devPtr1, sizeof(float) * 100);

  hipMemPoolProps poolProps = {};
  poolProps.allocType = hipMemAllocationTypePinned;
  poolProps.location.id = device;
  poolProps.location.type = hipMemLocationTypeDevice;

  hipMemPool_t pool;
  hipError_t err = hipMemPoolCreate(&pool, &poolProps);

  if (err != hipSuccess) {
    std::cerr << "Failed to create CUDA memory pool: "
              << hipGetErrorString(err) << std::endl;
    return 1;
  }

  void *devPtr = NULL;
  size_t size = 1024 * sizeof(int);
  hipError_t cudaStatus = hipMallocFromPoolAsync(&devPtr, size, pool, 0);
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipMallocFromPoolAsync failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    return 1;
  }

  int *hostPtr = new int[1024];
  hipMemcpy(hostPtr, devPtr, size, hipMemcpyDeviceToHost);

  hipFree(devPtr);
  hipMemPoolDestroy(pool);
  delete[] hostPtr;

  return 1;
}
